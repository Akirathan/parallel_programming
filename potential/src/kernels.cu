#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include "kernels.h"

// TODO: quickfix
#ifdef LOCAL
#define __global__
#endif

/*
 * Sample Kernel
 */
static __global__ void my_kernel(float *src)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	src[idx] += 1.0f;
}

static __global__ void array_add(const float *array_1, const float *array_2, float *dest, size_t size)
{
	size_t idx = threadIdx.x;
	assert(idx < size);
	dest[idx] = array_1[idx] + array_2[idx];
}

static __global__ void print_thread_idx(int *dest, size_t size)
{
	size_t idx = threadIdx.x;
	assert(idx < size);
	dest[idx] = 23;
}

static __global__ void compute_repulsive(const Point<double> *points, Point<double> *repulsive_forces_matrix,
        size_t points_size, double vertexRepulsion)
{
    const size_t row_size = points_size;

    size_t row = ((size_t)blockIdx.x * points_size / (size_t)blockDim.x) + threadIdx.x;
    size_t col = 0;

    assert(row < points_size && col < points_size);

    std::printf("Row = %d, col = %d\n", row, col);

    if (row < col) {
        double dx = points[row].x - points[col].x;
        double dy = points[row].y - points[col].y;
        double sqLen = dx*dx + dy*dy > (double)0.0001 ? dx*dx + dy*dy : (double)0.0001;
        double fact = vertexRepulsion / (sqLen * (double)std::sqrt(sqLen));	// mul factor
        dx *= fact;
        dy *= fact;

        repulsive_forces_matrix[row * row_size + col].x += dx;
        repulsive_forces_matrix[row * row_size + col].y += dy;

        repulsive_forces_matrix[col * row_size + row].x -= dx;
        repulsive_forces_matrix[col * row_size + row].y -= dy;
    }
}

static __global__ void compute_compulsive(const Point<double> *points, size_t points_size, const Edge<uint32_t> *edges,
        size_t edges_size, uint32_t length, Point<double> **forces, double edgeCompulsion)
{
    /*size_t i = threadIdx.x;
    size_t j = threadIdx.y;
    assert(i < points_size && j < points_size);

    double dx = points[i].x - points[j].x;
    double dy = points[i].y - points[j].y;
    double sqLen = dx*dx + dy*dy;
    double fact = (double)std::sqrt(sqLen) * edgeCompulsion / (double)(length);
    dx *= fact;
    dy *= fact;

    forces[j][i].x += dx;
    forces[j][i].y += dy;

    forces[i][j].x -= dx;
    forces[i][j].y -= dy;*/
}

/*
 * This is how a kernel call should be wrapped in a regular function call,
 * so it can be easilly used in cpp-only code.
 */
void run_my_kernel(float *src)
{
	my_kernel<<<64, 64>>>(src);
}

void run_array_add(const float *array_1, const float *array_2, float *dest, size_t size)
{
	assert(dest != nullptr);
	assert(size > 0 && size % 32 == 0);
	array_add<<<1, size>>>(array_1, array_2, dest, size);
}

void run_print_thread_idx(int *dest, size_t size)
{
	print_thread_idx<<<1, size>>>(dest, size);
}

void run_compute_repulsive(const Point<double> *points, size_t point_size, Point<double> *repulsive_forces_matrix,
        double vertexRepulsion)
{
    std::cout << "Running compute repulsive for (" << (unsigned)point_size << "," << (unsigned)point_size
              << "," << "1) thread dimensions" << std::endl;

    dim3 blocks;
    dim3 threads;
    size_t matrix_size = point_size * point_size;
    if (matrix_size > 1024) {

    }
    else {
        blocks = dim3{1, 1, 1};
        threads = dim3{(unsigned)point_size, (unsigned)point_size, 1};
    }

    compute_repulsive<<<blocks, threads>>>(points, repulsive_forces_matrix, point_size, vertexRepulsion);

    // Check if kernel was launched properly.
    CUCH(hipGetLastError());
}
