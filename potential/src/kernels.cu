#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include "kernels.h"

// TODO: quickfix
#ifdef LOCAL
#define __global__
#endif

/*
 * Sample Kernel
 */
static __global__ void my_kernel(float *src)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	src[idx] += 1.0f;
}

static __global__ void array_add(const float *array_1, const float *array_2, float *dest, size_t size)
{
	size_t idx = threadIdx.x;
	assert(idx < size);
	dest[idx] = array_1[idx] + array_2[idx];
}

static __global__ void print_thread_idx(int *dest, size_t size)
{
	size_t idx = threadIdx.x;
	assert(idx < size);
	dest[idx] = 23;
}

static __global__ void compute_repulsive(const Point<double> *points, Point<double> *repulsive_forces,
        size_t points_size, double vertexRepulsion)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    assert(i < points_size && j < points_size);

    if (i < j) {
        std::printf("\t\tKernel computing repulsive forces for i=%d, j=%d\n", i, j);
        double dx = points[i].x - points[j].x;
        double dy = points[i].y - points[j].y;
        double sqLen = dx*dx + dy*dy > (double)0.0001 ? dx*dx + dy*dy : (double)0.0001;
        double fact = vertexRepulsion / (sqLen * (double)std::sqrt(sqLen));	// mul factor
        dx *= fact;
        dy *= fact;

        atomicAdd(&repulsive_forces[i].x, dx);
        atomicAdd(&repulsive_forces[i].y, dy);
        atomicAdd(&repulsive_forces[j].x, -dx);
        atomicAdd(&repulsive_forces[j].y, -dy);
    }
}

static __global__ void compute_compulsive(const Point<double> *points, size_t points_size,
        const Edge<uint32_t> *edges, size_t edges_size,
        const uint32_t *lengths, size_t length_size,
        Point<double> *compulsive_forces, double edgeCompulsion)
{
    size_t edge_idx = blockIdx.x * blockDim.x + threadIdx.x;
    assert(edge_idx < edges_size);
    Edge<uint32_t> edge = edges[edge_idx];
    assert(edge.p1 < points_size && edge.p2 < points_size);

    double dx = points[edge.p2].x - points[edge.p1].x;
    double dy = points[edge.p2].y - points[edge.p1].y;
    double sqLen = dx*dx + dy*dy;
    double fact = (double)std::sqrt(sqLen) * edgeCompulsion / (double)(lengths[edge_idx]);
    dx *= fact;
    dy *= fact;

    atomicAdd(&compulsive_forces[edge.p1].x, dx);
    atomicAdd(&compulsive_forces[edge.p1].y, dy);
    atomicAdd(&compulsive_forces[edge.p2].x, -dx);
    atomicAdd(&compulsive_forces[edge.p2].y, -dy);
}

/*
 * This is how a kernel call should be wrapped in a regular function call,
 * so it can be easilly used in cpp-only code.
 */
void run_my_kernel(float *src)
{
	my_kernel<<<64, 64>>>(src);
}

void run_array_add(const float *array_1, const float *array_2, float *dest, size_t size)
{
	assert(dest != nullptr);
	assert(size > 0 && size % 32 == 0);
	array_add<<<1, size>>>(array_1, array_2, dest, size);
}

void run_print_thread_idx(int *dest, size_t size)
{
	print_thread_idx<<<1, size>>>(dest, size);
}

void run_compute_repulsive(const Point<double> *points, size_t point_size, Point<double> *repulsive_forces,
        double vertexRepulsion)
{
    assert(point_size % 2 == 0);

    dim3 blocks{1, 1, 1};
    dim3 threads{(unsigned)point_size, (unsigned)point_size, 1};
    while (threads.x * threads.y > 1024) {
        blocks.x *= 2;
        blocks.y *= 2;
        threads.x /= 2; // TODO: integer division?
        threads.y /= 2;
    }

    std::cout << "Running compute repulsive kernel for blocks_dim=(" << blocks.x << "," << blocks.y << ","
              << blocks.z << "), threads_dim=(" << threads.x << "," << threads.y << "," << threads.z
              << ")." << std::endl;
    compute_repulsive<<<blocks, threads>>>(points, repulsive_forces, point_size, vertexRepulsion);

    // Check if kernel was launched properly.
    CUCH(hipGetLastError());
}

void run_compute_compulsive(const Point<double> *points, size_t points_size,
                            const Edge<uint32_t> *edges, size_t edges_size,
                            const uint32_t *lengths, size_t lengths_size,
                            Point<double> *compulsive_forces_matrix, double edgeCompulsion)
{
    assert(edges_size % 2 == 0);

    dim3 blocks{1, 1, 1};
    dim3 threads{(unsigned)edges_size, 1, 1};
    while (threads.x > 1024) {
        blocks.x *= 2;
        threads.x /= 2;
    }
    std::cout << "Running compute compulsive kernel for blocks_dim=(" << blocks.x << "," << blocks.y << ","
              << blocks.z << "), threads_dim=(" << threads.x << "," << threads.y << "," << threads.z
              << ")." << std::endl;
    compute_compulsive<<<blocks, threads>>>
        (points, points_size, edges, edges_size, lengths, lengths_size, compulsive_forces_matrix, edgeCompulsion);
}
